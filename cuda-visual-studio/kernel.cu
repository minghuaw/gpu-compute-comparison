﻿
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <iostream>
#include <random>

#include "../cuda-kernel/kernels/common.h"
#include "../cuda-kernel/kernels/matmul.cuh"

using namespace common;

int main()
{
    matmul::hello<<<1, 1>>>();

    float* host_matrix_a, * host_matrix_b, * host_matrix_c;

    host_matrix_a = (float*)malloc(sizeof(float) * M * K);
    host_matrix_b = (float*)malloc(sizeof(float) * K * N);
    host_matrix_c = (float*)malloc(sizeof(float) * M * N);

    generate_random_matrix(host_matrix_a, M, K);
    generate_random_matrix(host_matrix_b, K, N);
    generate_zero_matrix(host_matrix_c, M, N);

    float* device_matrix_a, * device_matrix_b, * device_matrix_c;
    cudaCheck(hipMalloc((void**)&device_matrix_a, sizeof(float) * M * K));
    cudaCheck(hipMalloc((void**)&device_matrix_b, sizeof(float) * K * N));
    cudaCheck(hipMalloc((void**)&device_matrix_c, sizeof(float) * M * N));

    cudaCheck(hipMemcpy(device_matrix_a, host_matrix_a, sizeof(float) * M * K, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(device_matrix_b, host_matrix_b, sizeof(float) * K * N, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(device_matrix_c, host_matrix_c, sizeof(float) * M * N, hipMemcpyHostToDevice));
    cudaCheck(hipDeviceSynchronize());

    // Use naive implementation as a simple check for now
    float* host_expected, * device_expected;
    host_expected = (float*)malloc(sizeof(float) * M * N);
    cudaCheck(hipMalloc((void**)&device_expected, sizeof(float) * M * N));

    dim3 block_size(BM, BN, 1);
    dim3 grid_size(M / BM, N / BN, 1);
    matmul::naive<M, N, K><<<grid_size, block_size>>>(alpha, device_matrix_a, device_matrix_b, beta, device_expected);

    cudaCheck(hipMemcpy(host_expected, device_expected, sizeof(float) * M * N, hipMemcpyDeviceToHost));

    cudaCheck(hipDeviceSynchronize());

    float elapsed_time;
    hipEvent_t beg, end;
    cudaCheck(hipEventCreate(&beg));
    cudaCheck(hipEventCreate(&end));

    cudaCheck(hipEventRecord(beg));
    unsigned int repeats = 1;
    for (unsigned int i = 0; i < repeats; i++) {

        //block_size = dim3(BM, BN, 1);
        //grid_size = dim3(M / BM, N / BN, 1);
        //matmul::naive<M, N, K><<<grid_size, block_size>>>(alpha, device_matrix_a, device_matrix_b, beta, device_matrix_c);

        //block_size = dim3(BM, BN, 1);
        //grid_size = dim3(M / BM, N / BN, 1);
        //matmul::cache_blocking<M, N, K, BM, BN, BK><<<grid_size, block_size>>>(alpha, device_matrix_a, device_matrix_b, beta, device_matrix_c);

        block_size = dim3(BM / TM, BN, 1);
        grid_size = dim3(M / BM, N / BN, 1);
        matmul::block_tiling_1d<M, N, K, BM, BN, BK, TM><<<grid_size, block_size>>>(alpha, device_matrix_a, device_matrix_b, beta, device_matrix_c);

    }
    cudaCheck(hipEventRecord(end));

    cudaCheck(hipEventSynchronize(beg));
    cudaCheck(hipEventSynchronize(end));
    cudaCheck(hipEventElapsedTime(&elapsed_time, beg, end));

    printf("Average elapsed time: (%f) ms, performance: (%f) GFLOPS. size: (%d).\n",
        elapsed_time / (float)repeats, 2. * 1e-9 * (float)repeats * M * M * M / elapsed_time, M);

    cudaCheck(hipMemcpy(host_matrix_a, device_matrix_a, sizeof(float) * M * K, hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(host_matrix_a, device_matrix_a, sizeof(float) * K * N, hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(host_matrix_c, device_matrix_c, sizeof(float) * M * N, hipMemcpyDeviceToHost));
    cudaCheck(hipDeviceSynchronize());

    assert_eq(host_matrix_c, host_expected, M, N);

    free(host_matrix_a);
    free(host_matrix_b);
    free(host_matrix_c);
    cudaCheck(hipFree(device_matrix_a));
    cudaCheck(hipFree(device_matrix_b));
    cudaCheck(hipFree(device_matrix_c));

    return 0;
}

