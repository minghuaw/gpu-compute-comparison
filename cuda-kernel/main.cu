#include <iostream>
#include <random>
#include <chrono>

#include "kernels/common.h"
#include "kernels/matmul.cuh"
#include "kernels/ported.cuh"

#include "hip/hip_runtime.h"
#include "hipblas.h"

using namespace common;

void cublas_matmul(float *device_matrix_a, float *device_matrix_b, float *device_matrix_c) {
    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        printf("Create cublas handle error.\n");
        exit(EXIT_FAILURE);
    };

    //cublas列主序计算：https://www.cnblogs.com/cuancuancuanhao/p/7763256.html
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,N, M, K, &alpha, device_matrix_b, N, device_matrix_a, K, &beta, device_matrix_c, N);

    hipblasDestroy(handle);
}

int main() {
    matmul::hello<<<1, 1>>>();

    float *host_matrix_a, *host_matrix_b, *host_matrix_c;

    host_matrix_a = (float *)malloc(sizeof(float) * M * K);
    host_matrix_b = (float *)malloc(sizeof(float) * K * N);
    host_matrix_c = (float *)malloc(sizeof(float) * M * N);

    generate_random_matrix(host_matrix_a, M, K);
    generate_random_matrix(host_matrix_b, K, N);
    generate_zero_matrix(host_matrix_c, M, N);

    float *device_matrix_a, *device_matrix_b, *device_matrix_c;
    cudaCheck(hipMalloc((void **) &device_matrix_a, sizeof(float) * M * K));
    cudaCheck(hipMalloc((void **) &device_matrix_b, sizeof(float) * K * N));
    cudaCheck(hipMalloc((void **) &device_matrix_c, sizeof(float) * M * N));

    cudaCheck(hipMemcpy(device_matrix_a, host_matrix_a, sizeof(float) * M * K, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(device_matrix_b, host_matrix_b, sizeof(float) * K * N, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(device_matrix_c, host_matrix_c, sizeof(float) * M * N, hipMemcpyHostToDevice));
    cudaCheck(hipDeviceSynchronize());

    // Use naive implementation as a simple check for now
    float *host_expected, *device_expected;
    host_expected = (float *)malloc(sizeof(float) * M * N);
    cudaCheck(hipMalloc((void **)&device_expected, sizeof(float) * M * N));

    dim3 block_size(BM, BN, 1);
    dim3 grid_size(M / BM, N / BN, 1);
//    matmul::naive<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_expected);

    cublas_matmul(device_matrix_a, device_matrix_b, device_expected);
    cudaCheck(hipMemcpy(host_expected, device_expected, sizeof(float) * M * N, hipMemcpyDeviceToHost));

    cudaCheck(hipDeviceSynchronize());

    float elapsed_time;
    hipEvent_t beg, end;
    cudaCheck(hipEventCreate(&beg));
    cudaCheck(hipEventCreate(&end));

    cudaCheck(hipEventRecord(beg));
    uint repeats = 1;
    for (uint i = 0; i < repeats; i++) {
//        block_size = dim3(BM, BN, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        matmul::naive<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3(BM, BN, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        matmul::cache_blocking<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3(BM / TM, BN, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        matmul::block_tiling_1d<M, N, K, BM, BN, BK, TM><<<grid_size, block_size>>>(alpha, device_matrix_a, device_matrix_b, beta, device_matrix_c);

//        // A 2d block will give wrong result somehow
//        block_size = dim3(64, 1, 1);
//        grid_size = dim3(M / 64, N / 64, 1);
//        matmul::block_tiling_2d<M, N, K, 64, 64, 8, 8, 8><<<grid_size, block_size>>>(alpha, device_matrix_a, device_matrix_b, beta, device_matrix_c);

//        block_size = dim3(256, 1, 1);
//        grid_size = dim3(M / 128, N / 128, 1);
//        matmul::vectorize_block_tiling_2d<128, 128, 8, 8, 8><<<grid_size, block_size>>>(M, N, K, alpha, device_matrix_a, device_matrix_b, beta, device_matrix_c);

//        const uint K10_NUM_THREADS = 128;
//        const uint K10_BN = 128;
//        const uint K10_BM = 128;
//        const uint K10_BK = 16;
//        const uint K10_WN = 64;
//        const uint K10_WM = 64;
//        const uint K10_WNITER = 4;
//        const uint K10_TN = 4;
//        const uint K10_TM = 8;
//        dim3 blockDim(K10_NUM_THREADS);
//
//        constexpr uint NUM_WARPS = K10_NUM_THREADS / 32;
//
//        // warptile in threadblocktile
//        static_assert((K10_BN % K10_WN == 0) and (K10_BM % K10_WM == 0));
//        static_assert((K10_BN / K10_WN) * (K10_BM / K10_WM) == NUM_WARPS);
//
//        // threads in warpsubtile
//        static_assert((K10_WM * K10_WN) % (WARPSIZE * K10_TM * K10_TN * K10_WNITER) ==
//                      0);
//        constexpr uint K10_WMITER =
//                (K10_WM * K10_WN) / (32 * K10_TM * K10_TN * K10_WNITER);
//        // warpsubtile in warptile
//        static_assert((K10_WM % K10_WMITER == 0) and (K10_WN % K10_WNITER == 0));
//
//        static_assert((K10_NUM_THREADS * 4) % K10_BK == 0,
//                      "NUM_THREADS*4 must be multiple of K9_BK to avoid quantization "
//                      "issues during GMEM->SMEM tiling (loading only parts of the "
//                      "final row of Bs during each iteraion)");
//        static_assert((K10_NUM_THREADS * 4) % K10_BN == 0,
//                      "NUM_THREADS*4 must be multiple of K9_BN to avoid quantization "
//                      "issues during GMEM->SMEM tiling (loading only parts of the "
//                      "final row of As during each iteration)");
//        static_assert(K10_BN % (16 * K10_TN) == 0,
//                      "BN must be a multiple of 16*TN to avoid quantization effects");
//        static_assert(K10_BM % (16 * K10_TM) == 0,
//                      "BM must be a multiple of 16*TM to avoid quantization effects");
//        static_assert((K10_BM * K10_BK) % (4 * K10_NUM_THREADS) == 0,
//                      "BM*BK must be a multiple of 4*256 to vectorize loads");
//        static_assert((K10_BN * K10_BK) % (4 * K10_NUM_THREADS) == 0,
//                      "BN*BK must be a multiple of 4*256 to vectorize loads");
//
//        dim3 gridDim(CEIL_DIV(N, K10_BN), CEIL_DIV(M, K10_BM));
//        ported::sgemmWarptiling<K10_BM, K10_BN, K10_BK, K10_WM, K10_WN, K10_WNITER, K10_TM,
//                K10_TN, K10_NUM_THREADS>
//        <<<gridDim, blockDim>>>(M, N, K, alpha, device_matrix_a, device_matrix_b, beta, device_matrix_c);

    }
    cudaCheck(hipEventRecord(end));

    cudaCheck(hipEventSynchronize(beg));
    cudaCheck(hipEventSynchronize(end));
    cudaCheck(hipEventElapsedTime(&elapsed_time, beg, end));

    printf("Average elapsed time: (%f) ms, performance: (%f) GFLOPS. size: (%d).\n",
           elapsed_time / (float)repeats, 2. * 1e-9 * (float)repeats * M * M * M / elapsed_time, M);

    cudaCheck(hipMemcpy(host_matrix_a, device_matrix_a, sizeof(float) * M * K, hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(host_matrix_a, device_matrix_a, sizeof(float) * K * N, hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(host_matrix_c, device_matrix_c, sizeof(float) * M * N, hipMemcpyDeviceToHost));
    cudaCheck(hipDeviceSynchronize());

    assert_eq(host_matrix_c, host_expected, M, N);

    free(host_matrix_a);
    free(host_matrix_b);
    free(host_matrix_c);
    cudaCheck(hipFree(device_matrix_a));
    cudaCheck(hipFree(device_matrix_b));
    cudaCheck(hipFree(device_matrix_c));

    return 0;
}