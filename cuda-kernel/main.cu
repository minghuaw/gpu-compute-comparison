#include <iostream>
#include <random>
#include <chrono>

#include "kernels/matmul.cuh"
#include "kernels/ported_matmul.cuh"
#include "kernels/config.h"

#include "hip/hip_runtime.h"
#include "hipblas.h"

void cublas_sgemm(float *device_matrix_a, float *device_matrix_b, float *device_matrix_c) {
    using namespace config;

    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        printf("Create cublas handle error.\n");
        exit(EXIT_FAILURE);
    };

    //cublas列主序计算：https://www.cnblogs.com/cuancuancuanhao/p/7763256.html
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,N, M, K, &alpha, device_matrix_b, N, device_matrix_a, K, &beta, device_matrix_c, N);
    hipblasDestroy(handle);
}

/// Generate a contiguous random matrix in row major order
void generate_random_matrix(float *matrix, uint rows, uint cols) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(1.0, 5.0);

    for (uint i = 0; i < rows * cols; i++) {
        matrix[i] = dis(gen);
    }
}

void generate_zero_matrix(float *matrix, uint rows, uint cols) {
    for (uint i = 0; i < rows * cols; i++) {
        matrix[i] = 0;
    }
}

void cudaCheck(hipError_t error) {
    if (error != hipSuccess) {
        printf("%s", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

void assert_eq(float *value, float *expected, uint m, uint n) {
    for (uint i=0; i<m*n; i++) {
        if (value[i] != expected[i]) {
            printf("\x1B[31mError:\033[0m Value: %f is not equal to expected: %f at %d\n", value[i], expected[i], i);
            exit(EXIT_FAILURE);
        }
    }
}

int main() {
    using namespace config;

    float *host_matrix_a, *host_matrix_b, *host_matrix_c;

    host_matrix_a = (float *)malloc(sizeof(float) * M * K);
    host_matrix_b = (float *)malloc(sizeof(float) * K * N);
    host_matrix_c = (float *)malloc(sizeof(float) * M * N);

    generate_random_matrix(host_matrix_a, M, K);
    generate_random_matrix(host_matrix_b, K, N);
    generate_zero_matrix(host_matrix_c, M, N);

    float *device_matrix_a, *device_matrix_b, *device_matrix_c;
    cudaCheck(hipMalloc((void **) &device_matrix_a, sizeof(float) * M * K));
    cudaCheck(hipMalloc((void **) &device_matrix_b, sizeof(float) * K * N));
    cudaCheck(hipMalloc((void **) &device_matrix_c, sizeof(float) * M * N));

    cudaCheck(hipMemcpy(device_matrix_a, host_matrix_a, sizeof(float) * M * K, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(device_matrix_b, host_matrix_b, sizeof(float) * K * N, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(device_matrix_c, host_matrix_c, sizeof(float) * M * N, hipMemcpyHostToDevice));
    cudaCheck(hipDeviceSynchronize());

    // Use naive implementation as a simple check for now
    float *host_expected, *device_expected;
    host_expected = (float *)malloc(sizeof(float) * M * N);
    cudaCheck(hipMalloc((void **)&device_expected, sizeof(float) * M * N));

//    dim3 block_size(BM, BN, 1);
//    dim3 grid_size(M / BM, N / BN, 1);
//    matmul::naive<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_expected);

    dim3 block_size(BM, BN, 1);
    dim3 grid_size(M / BM, N / BN, 1);
    ported_matmul::naive<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_expected);

    cudaCheck(hipMemcpy(host_expected, device_expected, sizeof(float) * M * N, hipMemcpyDeviceToHost));
    printf("%f\n", host_expected[0]);

    cudaCheck(hipDeviceSynchronize());

    float elapsed_time;
    hipEvent_t beg, end;
    cudaCheck(hipEventCreate(&beg));
    cudaCheck(hipEventCreate(&end));

    cudaCheck(hipEventRecord(beg));
    uint repeats = 1;
    for (uint i = 0; i < repeats; i++) {
//        cublas_sgemm(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3(BM, BN, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        ported_matmul::naive<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3(BM * BN, 1, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        ported_matmul::mem_coalescing<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3(BM * BN, 1, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        ported_matmul::shaded_mem_block<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3((BM * BN) / TM, 1, 1);
//        grid_size = dim3(N / BN, M / BM, 1);
//        ported_matmul::block_tiling_1d<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3((BM * BN) / (TM * TN));
//        grid_size = dim3(N / BN, M / BM);
//        ported_matmul::block_tiling_2d<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3(BM, BN, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        matmul::naive<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3(BM, BN, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        matmul::cache_blocking<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);

//        block_size = dim3(BM / TM, BN / TN, 1);
//        grid_size = dim3(M / BM, N / BN, 1);
//        matmul::tiling<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b, device_matrix_c);
    }
    cudaCheck(hipEventRecord(end));

    cudaCheck(hipEventSynchronize(beg));
    cudaCheck(hipEventSynchronize(end));
    cudaCheck(hipEventElapsedTime(&elapsed_time, beg, end));

    printf("Average elapsed time: (%f) ms, performance: (%f) GFLOPS. size: (%d).\n",
           elapsed_time / (float)repeats, 2. * 1e-9 * (float)repeats * M * M * M / elapsed_time, M);

    cudaCheck(hipMemcpy(host_matrix_a, device_matrix_a, sizeof(float) * M * K, hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(host_matrix_a, device_matrix_a, sizeof(float) * K * N, hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(host_matrix_c, device_matrix_c, sizeof(float) * M * N, hipMemcpyDeviceToHost));
    cudaCheck(hipDeviceSynchronize());

    assert_eq(host_matrix_c, host_expected, M, N);

    free(host_matrix_a);
    free(host_matrix_b);
    free(host_matrix_c);
    cudaCheck(hipFree(device_matrix_a));
    cudaCheck(hipFree(device_matrix_b));
    cudaCheck(hipFree(device_matrix_c));

    return 0;
}